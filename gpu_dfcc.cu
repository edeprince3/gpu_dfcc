#include "hip/hip_runtime.h"
#include <libplugin/plugin.h>
#include <psi4-dec.h>
#include <libparallel/parallel.h>
#include <liboptions/liboptions.h>
#include <libmints/mints.h>
#include <libpsio/psio.hpp>
#include <../bin/fnocc/ccsd.h>
#include <../bin/fnocc/frozen_natural_orbitals.h>
#include "ccsd.h"

INIT_PLUGIN

using namespace boost;

namespace psi{ namespace fnocc {

extern "C" 
int read_options(std::string name, Options& options)
{
    if (name == "GPU_DFCC"|| options.read_globals()) {
      /*- do dgemm timings? -*/
      options.add_bool("DGEMM_TIMINGS",false);
      /*- max mapped memory (mb) -*/
      options.add_int("MAX_MAPPED_MEMORY",7000);
      /*- Override number of GPUs detected? -*/
      options.add_int("NUM_GPUS",0);
      /*- Do time each cc diagram? -*/
      options.add_bool("CC_TIMINGS",false);
      /*- Convergence for the CC energy.  Note that convergence is
          met only when E_CONVERGENCE and R_CONVERGENCE are satisfied. -*/
      options.add_double("E_CONVERGENCE", 1.0e-8);
      /*- Convergence criterion for Breuckner orbitals. The convergence
         is determined based on the largest $T_1$ amplitude. -*/
      options.add_double("BRUECKNER_ORBS_R_CONVERGENCE", 1e-5);
      /*- Maximum number of iterations for Brueckner orbitals optimization -*/
      options.add_int("BRUECKNER_MAXITER", 20);
      /*- Convergence for the CC amplitudes.  Note that convergence is
          met only when E_CONVERGENCE and R_CONVERGENCE are satisfied. -*/
      options.add_double("R_CONVERGENCE", 1.0e-7);
      /*- Maximum number of CC iterations -*/
      options.add_int("MAXITER", 100);
      /*- Desired number of DIIS vectors -*/
      options.add_int("DIIS_MAX_VECS", 8);
      /*- Do use low memory option for triples contribution? Note that this 
          option is enabled automatically if the memory requirements of the 
          conventional algorithm would exceed the available resources -*/
      options.add_bool("TRIPLES_LOW_MEMORY",false);
      /*- Do compute triples contribution? !expert -*/
      options.add_bool("COMPUTE_TRIPLES", true);
      /*- Do compute MP4 triples contribution? !expert -*/
      options.add_bool("COMPUTE_MP4_TRIPLES", false);
      /*- Do use MP2 NOs to truncate virtual space for QCISD/CCSD and (T)? -*/
      options.add_bool("NAT_ORBS", false);
      /*- Cutoff for occupation of MP2 NO orbitals in FNO-QCISD/CCSD(T)
          ( only valid if |fnocc__nat_orbs| = true ) -*/
      options.add_double("OCC_TOLERANCE", 1.0e-6);
      /*- Do SCS-MP2? -*/
      options.add_bool("SCS_MP2", false);
      /*- Do SCS-CCSD? -*/
      options.add_bool("SCS_CCSD", false);
      /*- Do SCS-CEPA? Note that the scaling factors will be identical
      to those for SCS-CCSD. -*/
      options.add_bool("SCS_CEPA", false);
      /*- Opposite-spin scaling factor for SCS-MP2 -*/
      options.add_double("MP2_SCALE_OS",1.20);
      /*- Same-spin scaling factor for SCS-MP2 -*/
      options.add_double("MP2_SCALE_SS",1.0/3.0);
      /*- Oppposite-spin scaling factor for SCS-CCSD -*/
      options.add_double("CC_SCALE_OS", 1.27);
      /*- Same-spin scaling factor for SCS-CCSD -*/
      options.add_double("CC_SCALE_SS",1.13);
      /*- Use packed storage for the (ac|bd) diagram? only valid in MO -*/
      options.add_bool("VABCD_PACKED",true);
      /*- do only evaluate mp2 energy? !expert -*/
      options.add_bool("RUN_MP2",false);
      /*- do only evaluate mp3 energy? !expert -*/
      options.add_bool("RUN_MP3",false);
      /*- do only evaluate mp4 energy? !expert -*/
      options.add_bool("RUN_MP3",false);
      /*- do only evaluate mp4 energy? !expert -*/
      options.add_bool("RUN_MP4",false);
      /*- do ccsd rather than qcisd? !expert -*/
      options.add_bool("RUN_CCSD",false);

      /*- Do use density fitting in CC? This keyword is used internally
          by the driver. Changing its value will have no effect on the 
          computation. -*/
      options.add_bool("DFCC",false);
      /*- Auxilliary basis for df-ccsd(t). -*/
      options.add_str("DF_BASIS_CC","");
      /*- tolerance for Cholesky decomposition of the ERI tensor -*/
      options.add_double("CHOLESKY_TOLERANCE",1.0e-4);

      /*- Is this a CEPA job? This parameter is used internally
      by the pythond driver.  Changing its value won't have any
      effect on the procedure. !expert -*/
      options.add_bool("RUN_CEPA",false);
      /*- Which coupled-pair method is called?  This parameter is
      used internally by the python driver.  Changing its value
      won't have any effect on the procedure. !expert -*/
      options.add_str("CEPA_LEVEL","CEPA(0)");
      /*- Compute the dipole moment? Note that dipole moments
      are only available in the FNOCC module for the ACPF, 
      AQCC, CISD, and CEPA(0) methods. -*/
      options.add_bool("DIPMOM",false);
      /*- Flag to exclude singly excited configurations from a 
      coupled-pair computation.  -*/
      options.add_bool("CEPA_NO_SINGLES",false);

    }

    return true;
}

extern "C" 
PsiReturnType gpu_dfcc(Options& options)
{
    boost::shared_ptr<Wavefunction> wfn;
    boost::shared_ptr<DFFrozenNO> fno(new DFFrozenNO(Process::environment.wavefunction(),options));
    fno->ThreeIndexIntegrals();
    if ( options.get_bool("NAT_ORBS") ) {
        fno->ComputeNaturalOrbitals();
        wfn = (boost::shared_ptr<Wavefunction>)fno;
    }else {
        wfn = Process::environment.wavefunction();
    }
    boost::shared_ptr<GPUDFCoupledCluster> ccsd (new GPUDFCoupledCluster(wfn,options));
    ccsd->compute_energy();

    return Success;
}

}} // End namespaces

